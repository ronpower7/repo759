#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <random>
#include "matmul.cuh"



__global__ void warm_up_kernel(){}



int main(int argc, char*argv[])
{
    hipEvent_t start;
    hipEvent_t stop;
    float ms1,ms2,ms3;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    int n = std::stoi(argv[1]);
    int block_dim = std::stoi(argv[2]);
    std::random_device entropy_source;
    std::mt19937_64 generator(entropy_source());
    std::uniform_int_distribution <int> dist1(-10,10);
    std::uniform_real_distribution <float> dist2(-10.0,10.0);
    std::uniform_real_distribution <double> dist3(-10.0,10.0);


    int*A1 = (int*)malloc(n*n*(sizeof(int)));
    int*B1 = (int*)malloc(n*n*(sizeof(int)));
    int*C1 = (int*)malloc(n*n*(sizeof(int)));
    int*d_A1,*d_B1,*d_C1;

    float*A2 = (float*)malloc(n*n*(sizeof(float)));
    float*B2 = (float*)malloc(n*n*(sizeof(float)));
    float*C2 = (float*)malloc(n*n*(sizeof(float)));
    float*d_A2,*d_B2,*d_C2;

    double*A3 = (double*)malloc(n*n*(sizeof(double)));
    double*B3 = (double*)malloc(n*n*(sizeof(double)));
    double*C3 = (double*)malloc(n*n*(sizeof(double)));
    double*d_A3,*d_B3,*d_C3;

    for(int i = 0; i < n*n; i++)
    {
        A1[i] = dist1(generator);
        B1[i] = dist1(generator);
        A2[i] = dist2(generator);
        B2[i] = dist2(generator);
        A3[i] = dist3(generator);
        B3[i] = dist3(generator);
    }
    hipMalloc((void**)&d_A1,sizeof(int) * n*n);
    hipMalloc((void**)&d_B1,sizeof(int) * n*n);
    hipMalloc((void**)&d_C1,sizeof(int) * n*n);

    hipMalloc((void**)&d_A2,sizeof(float) * n*n);
    hipMalloc((void**)&d_B2,sizeof(float) * n*n);
    hipMalloc((void**)&d_C2,sizeof(float) * n*n);

    hipMalloc((void**)&d_A3,sizeof(double) * n*n);
    hipMalloc((void**)&d_B3,sizeof(double) * n*n);
    hipMalloc((void**)&d_C3,sizeof(double) * n*n);


    hipMemcpy(d_A1,A1,sizeof(int)*n*n,hipMemcpyHostToDevice);
    hipMemcpy(d_B1,B1,sizeof(int)*n*n,hipMemcpyHostToDevice);
    hipMemcpy(d_C1,C1,sizeof(int)*n*n,hipMemcpyHostToDevice);

    hipMemcpy(d_A2,A2,sizeof(float)*n*n,hipMemcpyHostToDevice);
    hipMemcpy(d_B2,B2,sizeof(float)*n*n,hipMemcpyHostToDevice);
    hipMemcpy(d_C2,C2,sizeof(float)*n*n,hipMemcpyHostToDevice);

    hipMemcpy(d_A3,A3,sizeof(double)*n*n,hipMemcpyHostToDevice);
    hipMemcpy(d_B3,B3,sizeof(double)*n*n,hipMemcpyHostToDevice);
    hipMemcpy(d_C3,C3,sizeof(double)*n*n,hipMemcpyHostToDevice);

     hipEventRecord(start);
    matmul_1(d_A1,d_B1,d_C1,n,block_dim);
    hipEventRecord(stop);

    hipEventSynchronize(stop);
    hipMemcpy(C1,d_C1,sizeof(int)*n*n,hipMemcpyDeviceToHost);
    hipEventElapsedTime(&ms1, start, stop);

    hipEventRecord(start);
    matmul_2(d_A2,d_B2,d_C2,n,block_dim);
    hipEventRecord(stop);

    hipEventSynchronize(stop);
    hipMemcpy(C2,d_C2,sizeof(float)*n*n,hipMemcpyDeviceToHost);
    hipEventElapsedTime(&ms2, start, stop);

    hipEventRecord(start);
    matmul_3(d_A3,d_B3,d_C3,n,block_dim);
    hipEventRecord(stop);

    hipEventSynchronize(stop);
    hipMemcpy(C3,d_C3,sizeof(double)*n*n,hipMemcpyDeviceToHost);
    hipEventElapsedTime(&ms3, start, stop);
   
     std::cout<<C1[0]<<std::endl;
    std::cout<<C1[(n-1)*n + n-1];
    std::cout<<std::endl; 
    std::cout<<"Time ELapsed for int matmul "<<ms1;
    std::cout<<std::endl;
    std::cout<<std::endl;

    std::cout<<C2[0]<<std::endl;
    std::cout<<C2[(n-1)*n + n-1];
    std::cout<<std::endl; 
    std::cout<<"Time ELapsed for float matmul "<<ms2;
    std::cout<<std::endl;
    std::cout<<std::endl;

    std::cout<<C3[0]<<std::endl;
    std::cout<<C3[(n-1)*n + n-1];
    std::cout<<std::endl; 
    std::cout<<"Time ELapsed for double matmul "<<ms3;
    std::cout<<std::endl;
    std::cout<<std::endl;




    hipFree(d_A1);
    hipFree(d_B1);
    hipFree(d_C1);
    free(A1);
    free(B1);
    free(C1);
    hipFree(d_A2);
    hipFree(d_B2);
    hipFree(d_C2);
    free(A2);
    free(B2);
    free(C2);
    hipFree(d_A3);
    hipFree(d_B3);
    hipFree(d_C3);
    free(A3);
    free(B3);
    free(C3);


    return 0;
}

