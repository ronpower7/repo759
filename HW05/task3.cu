
#include <cstdio>
#include <hip/hip_runtime.h>
#include <cstdlib>
#include <iostream>
#include <random>
#include "vscale.cuh"



int main ( int argc , char* argv[] ) {
   std::random_device rd;  // a seed source for the random number engine
   std::mt19937 gen(rd()); // mersenne_twister_engine seeded with rd()
   std::uniform_real_distribution<> distribution_a(-10., 10.);
   std::uniform_real_distribution<> distribution_b(0., 1.);

   int N = std::atoi(argv[1]);
   
   float* h_a = new float[N] ;
   float* h_b = new float[N] ;

   for (int i = 0 ; i < N ; ++i) {
     h_a[i] = distribution_a(gen) ;
     h_b[i] = distribution_b(gen) ;
   }	   
   
   float* d_a  ;
   float* d_b  ;
   hipMalloc((void**) &d_a , sizeof(float) * N );
   hipMalloc((void**) &d_b , sizeof(float) * N );


   hipMemcpy(d_a,h_a,sizeof(float) * N,hipMemcpyHostToDevice);
   hipMemcpy(d_b,h_b,sizeof(float) * N,hipMemcpyHostToDevice);
   
   const int threadsPerBlock = std::atoi(argv[2]);
   const int blocksPerGrid   = ( N +  threadsPerBlock - 1 ) / threadsPerBlock ;

// Create CUDA events for timing
   hipEvent_t start, stop;
   hipEventCreate(&start);
   hipEventCreate(&stop);

    // Start recording
   hipEventRecord(start,0);

   vscale<<<blocksPerGrid,threadsPerBlock>>>(d_a,d_b,N);

    // Stop recording
   hipEventRecord(stop,0);
   hipEventSynchronize(stop);

    // Calculate elapsed time
   float elapsedTime = 0;
   hipEventElapsedTime(&elapsedTime, start, stop);


   hipMemcpy(h_b,d_b,sizeof(float) * N, hipMemcpyDeviceToHost);

   std::cout << elapsedTime << "ms" << std::endl ;
//   std::cout << h_b[0] << std::endl ;
 //  std::cout << h_b[N-1] << std::endl ;



   delete[] h_a;
   delete[] h_b;

   hipFree(d_a);
   hipFree(d_b);

   hipEventDestroy(start);
   hipEventDestroy(stop);
   return 0 ;


}	
