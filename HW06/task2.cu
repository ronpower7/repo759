#include <iostream>
#include <hip/hip_runtime.h>
#include "stencil.cuh"

#include <random>           // To generate random numbers
using namespace std;


int main(int argc, char *argv[])
{
    // command line arguments
    unsigned int n = std::stoi(argv[1]);                        // number of elements
    unsigned int R = std::stoi(argv[2]);
    unsigned int threads_per_block = std::stoi(argv[3]);        // threads per block
    
    // declarations for calculating time
    hipEvent_t start;
    hipEvent_t stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // random number generation
    std::random_device entropy_source;
    std::mt19937_64 generator(entropy_source());
    std::uniform_real_distribution<float> dist1(-1.0, 1.0);
    std::uniform_real_distribution<float> dist2(-1.0, 1.0);


    // host arrays 
    float *image= (float*)malloc(n*(sizeof(float)));
    float *output= (float*)malloc(n*(sizeof(float)));
    float *mask= (float*)malloc((2*R+1)*(sizeof(float)));

    for (size_t i = 0; i < n; i++)
    {
        image[i] = dist1(generator);
    }

    for (size_t i = 0; i < (2*R+1); i++)
    {
        mask[i] = dist2(generator);
    }

    // device arrays
    float *d_image, *d_mask, *d_output;

    // allocate memory on the device (GPU)
    hipMalloc((void **)&d_image, n * sizeof(float));
    hipMalloc((void **)&d_mask, (2*R+1) * sizeof(float));
    hipMalloc((void **)&d_output, n * sizeof(float));

    // copy data into device blocks
    hipMemcpy(d_image, image, n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_mask, mask, (2*R+1) * sizeof(float), hipMemcpyHostToDevice);


    // #############################################
    hipEventRecord(start);
    stencil(d_image, d_mask, d_output, n, R, threads_per_block);
    hipEventRecord(stop);
    // #############################################
    
    hipEventSynchronize(stop);
    float elapsedtime;                                                  
    hipEventElapsedTime(&elapsedtime, start, stop);

    hipMemcpy(output, d_output, n * sizeof(float), hipMemcpyDeviceToHost);




    std::cout << output[n - 1] << std::endl;
    std::cout << elapsedtime << "ms"  << std::endl;

    hipEventDestroy(start);
    hipEventDestroy(stop);

    // deallocate memory from Host and Device
    hipFree(d_image);
    hipFree(d_mask);
    hipFree(d_output);
    free(image);
    free(mask);
    free(output);


    return 0;
    
}

